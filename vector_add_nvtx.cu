#include "hip/hip_runtime.h"
#include <stdio.h>
#include <nvToolsExt.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}

int main() {
    const int N = 5;
    int ha[N] = {1,2,3,4,5}, hb[N] = {10,20,30,40,50}, hc[N];
    int *da, *db, *dc;

    // -------------------- Malloc --------------------
    nvtxRangePushA("cudaMalloc1");
    hipMalloc((void**)&da, N * sizeof(int));
    nvtxRangePop();
    
    nvtxRangePushA("cudaMalloc2");
    hipMalloc((void**)&db, N * sizeof(int));
    nvtxRangePop();

    nvtxRangePushA("cudaMalloc3");
    hipMalloc((void**)&dc, N * sizeof(int));
    nvtxRangePop();

    // -------------------- H2D Memcpy --------------------
    nvtxRangePushA("H2D memcpy");
    hipMemcpy(da, ha, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, N * sizeof(int), hipMemcpyHostToDevice);
    nvtxRangePop();

    // -------------------- Kernel --------------------
    nvtxRangePushA("Kernel launch");
    add<<<1, N>>>(da, db, dc);
    hipDeviceSynchronize();  // 커널 실행 기다림
    nvtxRangePop();

    // -------------------- D2H Memcpy --------------------
    nvtxRangePushA("D2H memcpy");
    hipMemcpy(hc, dc, N * sizeof(int), hipMemcpyDeviceToHost);
    nvtxRangePop();

    // -------------------- 결과 출력 --------------------
    for (int i = 0; i < N; i++)
        printf("%d + %d = %d\n", ha[i], hb[i], hc[i]);

    // -------------------- Free --------------------
    nvtxRangePushA("hipFree");
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    nvtxRangePop();

    return 0;
}

